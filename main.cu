#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <time.h>
#include <cmath>

// if N >= 65 535 then error


#include <vector>
#include <stdio.h>



__device__ int division_f(int a, int b, int f){
        int result = 0;
        int count = 1;
        while (count <= f){
            if ((b*count)%f == a){
                return count;
            }
            count += 1;
        }
        return -1;
    }

 __device__ int getsize(int *ptr, int field){ //works fine
         //std::cout<<ptr[0]<<ptr[1]<<ptr[2]<<ptr[3]<<std::endl;
        int result = 0;

        while (ptr[result] != field+1 ) 
        {
        result++;
        }
        return result+1;
    }

__device__ int diff_f(int a, int b, int f){
        if ((a-b)<0){
            return a-b+f;
        }
        else{
            return a-b;
        }
    }

__device__ int sum_f(int a, int b, int f){
        if ((a+b)>=f){
            return (a+b)-f;
        }
        else{
            return a+b;
        }
    }




__device__ int division(int* A_in, int* B_in, int f, int* answer){
        const int a_len = getsize(A_in, f);
        const int b_len = getsize(B_in, f);
        int* B = new int[b_len];
        for (int i=0;i<b_len;i++) B[i] = B_in[i];

        int result_flag = 0;
        int* result = new int[a_len]; 
        for (int i=0; i<a_len; i++) {
            result[result_flag] = A_in[i]; 
            result_flag++;
            }
        int code = 0;
        
        while (true) {
            int cursorIndex = 1;
            if (getsize(result, f) < getsize(B, f)) {
                break;
            }
            else{

                int res_size = getsize(result,f);
                int* A = new int[res_size]; 
                int* tmp; 
                tmp=result; 
                for (int i = 0; i<res_size; i++)  
                {
                    A[i] = *tmp; 
                    tmp++;
                }  

                result_flag = 0;
                for (int r=0; r<res_size;r++){
                    result[r] = f+1;
                }
                int c = division_f(A[0], B[0], f);

                if (c==-1){
                    code = -1;
                    break;
                }
                int flag = 0;
                for (int i = 1; i<getsize(B, f)-1; i++){ 
                    cursorIndex += 1;
                    if (flag == 0) {
                        if (A[i] == c*B[i]){
                            continue;
                        }
                        else {
                            flag = 1;
                            result[result_flag] = diff_f(A[i]%f, c*B[i]%f, f); 
                            result_flag++;
                        }
                    }
                    else {
                        result[result_flag] = diff_f(A[i]%f, c*B[i]%f, f); 
                        result_flag++;
                    }
                    
                }
                if (cursorIndex < getsize(A, f)-1){
                    for (cursorIndex; cursorIndex<getsize(A, f); cursorIndex++){
                        result[result_flag] = A[cursorIndex]; 
                        result_flag++;
                    }
                }
            }

        }
        int i =0;

        if ((getsize(result,f) == 2) && result[0] == 0){
            
                answer[0] = f+1;
            
        }
        else{
                for (int an=0; an<getsize(result,f); an++){
                    answer[an] = result[an];
                }
            }
        return 0;
    }
    


int strInt(std::string inputString){
    int result = 0;
    int razr = 1;
    for (int i = inputString.length()-1; i>=0; i--){
        int f  = inputString[i]-'0';
        result = result + f*razr;
        razr = razr*10;
    }
    return result;
}

__device__ bool bigger(int* A, int* B, int f){

    if (getsize(A,f) > getsize(B,f)) return true;
    if (getsize(A,f) < getsize(B,f)) return false;
    for (int i=0; i<getsize(A,f); i++){
        if (A[i] > B[i]) return true;
        if (A[i] < B[i]) return false;
    }
}


class Polinom {     
    public:
    std::vector<int> koefs;
   
    Polinom(std::vector<int> input_koefs){
        for (int i: input_koefs)
            this->koefs.push_back(i);
    }
    void printPolinom(){
        for (int i: koefs){
            std::cout<<i;
        }
        std::cout<<std::endl;
    }

    std::vector<int> getPolinom(){
        return koefs;
    }

    int getLength(){
        return koefs.size();
    }

    int division_f(int a, int b, int f){
        int result = 0;
        int count = 1;
        while (count <= f){
            if ((b*count)%f == a){
                return count;
            }
            count += 1;
        }
        return -1;
    }

    int diff_f(int a, int b, int f){
        if ((a-b)<0){
            return a-b+f;
        }
        else{
            return a-b;
        }
    }

    int sum_f(int a, int b, int f){
        if ((a+b)>=f){
            return (a+b)-f;
        }
        else{
            return a+b;
        }
    }

    Polinom sum (Polinom B, int f){
        std::vector<int> result = koefs;
        if (koefs.size() > B.koefs.size()){
           int dif = koefs.size() - B.koefs.size();
           for (int i =0; i<B.koefs.size(); i++){
                result[dif+i] = result[dif+i] + B.koefs[i];
           }
        }
        else{
        result = B.koefs;
        int dif = B.koefs.size() - koefs.size();
           for (int i =0; i<koefs.size(); i++){
                result[dif+i] = result[dif+i] + koefs[i];
           }
        }
        return Polinom(result);
    }

    bool bigger(Polinom B, int f) {
        if (koefs.size() == B.getLength()){
            for (int i=0; i<B.getLength(); i++){
                if (koefs[i] == B.koefs[i]) {
                    continue;
                }
                else{
                    if (koefs[i]%f > B.koefs[i]%f){
                        return true;
                    }
                    else{
                        return false;
                    }
                }
            }
        }
        else {
            if(koefs.size() > B.getLength()){
                return true;
            }
            else {
                return false;
            }
        }
    return true;
    }

    Polinom division(Polinom B, int f){
        Polinom A(koefs);
        std::vector<int> result=koefs;
        int code = 0;
        while (true) {
            int cursorIndex = 1;
            if (result.size() < B.koefs.size()) {
                break;
            }
            else{
                A.koefs = result;
                result={};
                int c = division_f(A.koefs[0], B.koefs[0], f);
                if (c==-1){
                    code = -1;
                    break;
                }
                int flag = 0;
                for (int i = 1; i<B.koefs.size(); i++){
                    cursorIndex += 1;
                    if (flag == 0) {
                        if (A.koefs[i] == c*B.koefs[i]){
                            continue;
                        }
                        else {
                            flag = 1;
                            result.push_back(diff_f(A.koefs[i]%f, c*B.koefs[i]%f, f));
                        }
                    }
                    else {
                        result.push_back(diff_f(A.koefs[i]%f, c*B.koefs[i]%f, f));
                    }
                    
                }
                if (cursorIndex < A.getLength()){
                      for (cursorIndex; cursorIndex<A.getLength(); cursorIndex++){
                        result.push_back(A.koefs[cursorIndex]);
                    }
                }
            }

        }
        return Polinom(result);
    }
    

  
};



Polinom recAlg(Polinom A, Polinom B, int f){
    if ( A.getLength() != 0 && B.getLength() != 0){
        if ((A.koefs[0] == 0) || (B.koefs[0] == 0)){
            return A.sum(B, f);
        }
        if (A.bigger(B, f)) {
            return recAlg(A.division(B,f), B, f);
        }
        else{
            return recAlg(B.division(A,f), A, f);
        }
    }
    return A.sum(B, f);
}



__device__ void recAlg_norm(int* A_in, int* B_in, int f, int* nod){
        int a_len = getsize(A_in, f);
        int b_len = getsize(B_in, f);
        int* B = new int[b_len];
        int* A = new int[a_len];
        for (int i=0;i<b_len;i++) B[i] = B_in[i];
        for (int i=0;i<a_len;i++) A[i] = A_in[i];
    
    while (true){
    if (getsize(A, f) != 1 && getsize(B, f) != 1){
            if (A[0] == f+1) {
                for (int an=0; an<getsize(B,f); an++){
                    nod[an] = B[an];
                }
                break;
                }
                if (B[0] == f+1) {
                for (int an=0; an<getsize(A,f); an++){
                    nod[an] = A[an];
                }
                break;
                }
        
        if (bigger(A, B, f)) {

            int* C = new int[getsize(A, f)];
            division(A,B,f, C);
            int c_res = getsize(C,f);
            for (int c_size=0;c_size<c_res; c_size++) A[c_size] = C[c_size];


        }
        else{
            int* C = new int[getsize(B, f)];
            division(B,A,f, C);
            for (int c_size=0;c_size<getsize(C,f); c_size++) B[c_size] = C[c_size];
        }
    }
    else{
        if (A[0] == f+1) {
            for (int an=0; an<getsize(B,f); an++){
                    nod[an] = B[an];
            }
            }
        if (B[0] == f+1) {
            for (int an=0; an<getsize(A,f); an++){
                    nod[an] = A[an];
            }
            }
            break;
    }
    }
    }


__global__ void kernel_counter(int* ptr, int* input_array, int field, int count_polinoms, int size_array, int point_of_answer){

    int tid = threadIdx.x+blockIdx.x*250;
    if(tid<size_array) {
        input_array[tid]=ptr[tid];

    }
    if (tid < count_polinoms){
    while (input_array[point_of_answer-1] == field)
    {
    int i = 0;
    int counter_f = 0;
    int start_point = 0;
    int first_pol_size=0;
    int middle_point = 0;
    int end_point = 0;
    int second_pol_size = 0;
    int answer_point = 0;
    if (counter_f < tid+count_polinoms){
            while (counter_f < 2*tid){
                if (ptr[i] == field + 1){
                    counter_f++;
                }
                i++;
            }
            start_point = i;
        
        
            while (counter_f < 2*tid+1){
                if (ptr[i] == field + 1){
                    counter_f++;
                }
                i++;
                first_pol_size++;
            }
            middle_point = i;
        
            while (counter_f < 2*tid+2){
                if (ptr[i] == field + 1){
                    counter_f++;
                }
                i++;
                second_pol_size++;
            }
            end_point = i;


            while (counter_f < tid+count_polinoms){
                if (ptr[i] == field + 1){
                    counter_f++;
                }
                i++;
            }
            answer_point = i;
    }


        if (input_array[start_point] == field){
          
        continue;
        }

        if (input_array[middle_point] == field){
          
            continue;
        }

    int* A = new int[first_pol_size];
    int* B = new int[second_pol_size];
    int build_counter = start_point;
    int build_counter_a = 0;
    int build_counter_b = 0;
    while (build_counter < end_point){
        while (build_counter < middle_point){
            A[build_counter_a] = input_array[build_counter];
            build_counter++;
            build_counter_a++;
        }
        B[build_counter_b] = input_array[build_counter];
        build_counter++;
        build_counter_b++;
    }

    int result_value = getsize(A,field);
    if (result_value<getsize(B,field))
        result_value = getsize(B,field);
    int* nod = new int[result_value];
    recAlg_norm(A, B, field, nod);
    if (getsize(nod,field) == 2 && nod[0]==1){
      input_array[point_of_answer-1] =1;
    }
    for (int num =0; num<getsize(nod, field); num++){
        input_array[answer_point] = nod[num];
        answer_point++;
    }
      }
    }
    }
  



int main(){
std::vector<Polinom> unsortedPolinoms;
    std::vector<Polinom> polinoms_1;
    std::vector<Polinom> polinoms;

    int field = 0;
    std::string fieldCin;

    std::cout << "enter field\n";
    std::cin >> fieldCin;

    field = pow(2,strInt(fieldCin));

    int polinomCount;
    std::cout << "enter polinoms, press 'e' when u want to stop\n";

    int flag_field = 0;

    while (true) {
        std::string g;
        std::cin >> g;

        if (g == "e") {
            break;
        }

        std::vector<int> test_vec;
        for (int letter = 0; letter < g.length(); letter++) {
            int digit;
            digit = g[letter] - '0';
            test_vec.push_back(digit);
        }
        unsortedPolinoms.push_back(Polinom(test_vec));
    }
    
    

    int maxLength = 0; 
    
    for (Polinom i: unsortedPolinoms){
        if (maxLength < i.getLength()){
            maxLength = i.getLength();
        }
    }
    
    for (int j=0; j<=maxLength; j++){
        for (Polinom p: unsortedPolinoms){
            if (p.getLength() == j){
                polinoms_1.push_back(p);
       //         p.printPolinom();
            }
        }
    }

    int pol_count = polinoms_1.size()-1;

    int cor_pol = 0;

    while (cor_pol < pol_count-cor_pol){
            if (cor_pol == pol_count - cor_pol) polinoms.push_back(polinoms_1[cor_pol]);
            polinoms.push_back(polinoms_1[cor_pol]);
            polinoms.push_back(polinoms_1[pol_count-cor_pol]);
            cor_pol++;
        
    }

    std::cout << "sort by division result: " << "\n";
    for (Polinom p:polinoms) p.printPolinom();

    int count_symbols = 0;


    int count_polinoms = polinoms.size();

    for (int p_s=0; p_s<polinoms.size();p_s++){
        for (int pol_s=0; pol_s<polinoms[p_s].koefs.size();pol_s++){
            count_symbols ++;
        }
    }

    int all_symbols_in_result_array = (count_polinoms*2-1) + count_symbols + maxLength*(count_polinoms-1);

    int input_array[1000];


    int great_counter = 0;

    for (int p_s=0; p_s<polinoms.size();p_s++){
        for (int pol_s=0; pol_s<polinoms[p_s].koefs.size();pol_s++){
            input_array[great_counter] = polinoms[p_s].koefs[pol_s];
            great_counter++;
        }
        input_array[great_counter] = field+1;
        great_counter++;
    }

    for (int extra_pol =0; extra_pol<count_polinoms-1;extra_pol++){
        for (int field_number=0; field_number<maxLength; field_number++){
            input_array[great_counter] = field;
            great_counter++;
        }
        input_array[great_counter] = field+1;
        great_counter++;
    }


    std::cout << "max length: "<< maxLength << " polinom count: " << count_polinoms << "\n";

    int *input_for_kernel;
    int *result_after_sorting;

    int sort_result[1000];

  hipMalloc((void**)&input_for_kernel, great_counter * sizeof(int));
  hipMalloc((void**)&result_after_sorting, great_counter * sizeof(int));


hipMemcpy(input_for_kernel, input_array, great_counter * sizeof(int), hipMemcpyHostToDevice);

hipEvent_t start, stop;
    float gpuTime = 0.0;

    hipEventCreate( &start );
    hipEventCreate( &stop );
    hipEventRecord( start, 0 );

    

  kernel_counter <<<30, 250>>> (input_for_kernel, result_after_sorting, field, count_polinoms, great_counter, great_counter-maxLength);

hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    hipEventElapsedTime( &gpuTime, start, stop );
    printf("viduxa time\n", gpuTime);

    hipEventDestroy( start );
    hipEventDestroy( stop );


  hipMemcpy(sort_result, result_after_sorting, great_counter * sizeof(int), hipMemcpyDeviceToHost);

  hipFree(input_for_kernel);
  hipFree(result_after_sorting);

  std::cout<<"NOD = ";
  for (int i =great_counter-(maxLength+1); i<great_counter;i++){
    if (sort_result[i] == field+1) break;
    if (sort_result[i] == field) break;
    std::cout<<sort_result[i];
  }

  std::cout << "\n";

  system("pause");

  return 1;
}
